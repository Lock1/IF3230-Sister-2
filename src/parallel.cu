#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>

#include <stdio.h>

#define BLOCK_LENGTH 16

#include "serial_lib.hpp"
using namespace std;

__device__ int supression_op(Matrix &kernel, Matrix &target, int row, int col) {
	int intermediate_sum = 0;
	for (int i = 0; i < kernel.row_eff; i++)
		for (int j = 0; j < kernel.col_eff; j++)
			intermediate_sum += kernel.mat[i][j] * target.mat[row + i][col + j];

	return intermediate_sum;
}

__global__ void convolution(Matrix *d_kernel, Matrix *d_target, Matrix *d_result) {
    __shared__ Matrix kernel;
    kernel = *d_kernel;
    int target_row = d_target[0].row_eff;
    int target_col = d_target[0].col_eff;
    __syncthreads();

    for (int i = threadIdx.x; i < target_row; i += BLOCK_LENGTH) {
        for (int j = threadIdx.y; j < target_col; j += BLOCK_LENGTH) {
            int inter_sum = supression_op(kernel, d_target[blockIdx.x], i, j);
            d_result[blockIdx.x].mat[i][j] = inter_sum;
        }
    }
}

int cmpfunc(void const *a, void const *b) {
   return *(int*)a - *(int*)b;
}

void compute_convolution(ifstream &fs) {
    int kernel_row, kernel_col, target_row, target_col, num_targets;
    Matrix kernel;

    // File stream processing
    fs >> kernel_row >> kernel_col;
    kernel = input_matrix(fs, kernel_row, kernel_col);
    fs >> num_targets >> target_row >> target_col;

    Matrix *target_container = new Matrix[num_targets];
    for (int i = 0; i < num_targets; i++)
        target_container[i] = input_matrix(fs, target_row, target_col);

    Matrix *result_container = new Matrix[num_targets];
    int res_row = target_row - kernel_row + 1;
    int res_col = target_col - kernel_col + 1;
    for (int i = 0; i < num_targets; i++)
        init_matrix(result_container[i], res_row, res_col);

    // Host to device memory copy
    Matrix *d_kernel;
    hipMalloc((void **) &d_kernel, sizeof(Matrix));
    hipMemcpy(d_kernel, &kernel, sizeof(Matrix), hipMemcpyHostToDevice);

    Matrix *d_target;
    hipMalloc((void **) &d_target, sizeof(Matrix)*num_targets);
    hipMemcpy(d_target, target_container, sizeof(Matrix)*num_targets, hipMemcpyHostToDevice);

    Matrix *d_result;
    hipMalloc((void **) &d_result, sizeof(Matrix)*num_targets);
    hipMemcpy(d_result, result_container, sizeof(Matrix)*num_targets, hipMemcpyHostToDevice);

    // Device execution
    dim3 gridDim(num_targets);
    dim3 blockDim(BLOCK_LENGTH, BLOCK_LENGTH);
    convolution<<<gridDim, blockDim>>>(d_kernel, d_target, d_result);
    hipDeviceSynchronize();

    // Result processing
    hipError_t err = hipMemcpy(result_container, d_result, sizeof(Matrix)*num_targets, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
        printf("CUDA error copying to Host: %s\n", hipGetErrorString(err));

    int *matrix_ranges = new int[num_targets];
    for (int i = 0; i < num_targets; i++)
        matrix_ranges[i] = get_matrix_datarange(result_container[i]);

    qsort(matrix_ranges, num_targets, sizeof(int), cmpfunc);

    int median       = get_median(matrix_ranges, num_targets);
    int floored_mean = get_floored_mean(matrix_ranges, num_targets);

    printf("%d\n%d\n%d\n%d\n",
            matrix_ranges[0],
            matrix_ranges[num_targets - 1],
            median,
            floored_mean);


    // Release memory
    delete target_container;
    delete result_container;
    hipFree(d_kernel);
    hipFree(d_target);
    hipFree(d_result);
}


int main(int argc, char const *argv[]) {
    ifstream fs(argv[1]);

    if (argc > 1 && fs.is_open()) {
        compute_convolution(fs);
        fs.close();
    }
    else {
        cout << "parallel: Failed to open file\n";
        exit(1);
    }

    return 0;
}
