#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>

#include <stdio.h>

#include "serial_lib.hpp"
using namespace std;

__global__ void convolution(Matrix *d_kernel) {
    __shared__ Matrix kernel;
    kernel = *d_kernel;
    __syncthreads();

    // printf("<%d, %d> %d\n", threadIdx.x, threadIdx.y, kernel.mat[threadIdx.x][threadIdx.y]);
}

void compute_convolution(ifstream &fs) {
    int kernel_row, kernel_col, target_row, target_col, num_targets;
    Matrix kernel;

    fs >> kernel_row >> kernel_col;
    kernel = input_matrix(fs, kernel_row, kernel_col);

    fs >> num_targets >> target_row >> target_col;

    Matrix *d_kernel;
    hipMalloc((void **) &d_kernel, sizeof(Matrix));
    hipMemcpy(d_kernel, &kernel, sizeof(Matrix), hipMemcpyHostToDevice);

    dim3 gridDim(1);
    dim3 blockDim(16, 16);
    convolution<<<gridDim, blockDim>>>(d_kernel);
    hipDeviceSynchronize();
}


int main(int argc, char const *argv[]) {
    ifstream fs(argv[1]);

    if (argc > 1 && fs.is_open()) {
        compute_convolution(fs);
        fs.close();
    }
    else {
        cout << "parallel: Failed to open file\n";
        exit(1);
    }

    return 0;
}




// __global__ void add(int *d_a, int *d_b, int *d_c) {
//   *d_c = *d_a + *d_b;
// }

// int main() {
//   int a, b, c; // host copies of variables a, b & c
//   int *d_a, *d_b, *d_c; // device copies of variables a, b & c
//
//   int size = sizeof(int); // Allocate space for device copies of a, b, c
//   hipMalloc((void **)&d_a, size);
//   hipMalloc((void **)&d_b, size);
//   hipMalloc((void **)&d_c, size);
//
//   // Setup input values
//   c = 0;
//   a = 10;
//   b = 11;
//
//   // Copy inputs to device
//   hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
//   hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
//
//   // Launch add() kernel on GPU
//   add<<<1,1>>>(d_a, d_b, d_c);
//
//   // Copy result back to host
//   hipError_t err = hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
//   if(err!=hipSuccess) {
//     printf("CUDA error copying to Host: %s\n", hipGetErrorString(err));
//   }
//
//   printf("result is %d\n", c);
//
//   if (c != 21) {
//     printf("Something wrong.\n");
//   }
//
//   // Cleanup
//   hipFree(d_a);
//   hipFree(d_b);
//   hipFree(d_c);
//   return 0;
// }
